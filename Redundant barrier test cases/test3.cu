
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void test(int *data) {
    // give for loop a with redundant barrier
        for (int i = 0; i < 256; i++) {
            __syncthreads();
            data[i] = i;
            __syncthreads();
        }
        // give for loop a without redundant barrier
        for (int i = 0; i < 256; i++) {
            data[i] = i;
        }
        // give for loop b with redundant barrier
        for (int i = 0; i < 256; i++) {
            __syncthreads();
            data[i] = i;
            __syncthreads();
        }
        // give for loop b without redundant barrier
        for (int i = 0; i < 256; i++) {
            data[i] = i;
        }
        
    }

int main() {
    int *d_data;
    hipMalloc(&d_data, sizeof(int) * 256);
    test<<<1, 256>>>(d_data);
    hipDeviceSynchronize();
    hipFree(d_data);
    return 0;
}

