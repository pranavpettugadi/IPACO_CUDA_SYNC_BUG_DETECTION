#include <iostream>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

__global__ void vectorAdd(const float* A, const float* B, float* C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int a = 3, b = 5, c = 4, d = 1;
    if(a > 3) {
        b = 2;
    }
    else {
        b = 10;
    }
    int x = (b*i + 4) / d;
    if(x < (2*i + 30) * 2) {
        __syncthreads();
    }
}

int main() {
    int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy input data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    bool success = true;
    for (int i = 0; i < N; ++i) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            success = false;
            break;
        }
    }

    std::cout << (success ? "Success!" : "Error!") << std::endl;

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
