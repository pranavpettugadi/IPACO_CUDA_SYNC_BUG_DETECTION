
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void test(int *data) {
    int tid = threadIdx.x;
    int a,b=0,c=tid;
    if(tid == 0) {
        a=10;
    }
    else {
        a=20;
    }
    data[a*tid] = tid;
    __syncthreads();  // Redundant: no access after this depends on data written before
    data[tid] = tid;
        
}


int main() {
    int *d_data;
    hipMalloc(&d_data, sizeof(int) * 256);
    test<<<1, 256>>>(d_data);
    hipDeviceSynchronize();
    hipFree(d_data);
    return 0;
}
