#include <hip/hip_runtime.h>
#include <iostream>

__global__ void myKernel(int *a, int x, int y) {
    int tid = threadIdx.x;
    a[tid] = x;
    __syncthreads();
    a[tid + 1] = y; // Not a Data Race
    a[tid] = x; // Data Race
}

int main() {
    const int N = 10;
    int *d_a, h_a[N];

    // Allocate memory on the device
    hipMalloc(&d_a, N * sizeof(int));

    // Launch kernel with 1 block and 1 thread
    myKernel<<<1, 1>>>(d_a, 5, 10);

    // Copy results back to host
    hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the first few elements
    for (int i = 0; i < 4; ++i)
        std::cout << "a[" << i << "] = " << h_a[i] << std::endl;

    // Free device memory
    hipFree(d_a);
    return 0;
}