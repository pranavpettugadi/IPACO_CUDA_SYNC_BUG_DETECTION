
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void test0(int *data) {
    int tid = threadIdx.x;
    __syncthreads();  // Redundant: no access after this depends on data written before
    data[tid] = tid;
    
}

__global__ void test1(int *data) {
    int tid = threadIdx.x;
    data[tid] = tid;
    __syncthreads();  // Redundant: no access after this depends on data written before
    int x = tid + 1;
}

__global__ void test2(int *data) {
    int tid = threadIdx.x;
    data[tid] = tid;
    __syncthreads();  // Necessary: next line reads from data written by other threads
    int val = data[(tid + 1) %256];  // Accesses data written by other threads
}

__global__ void test3(int *data) {
    int tid = threadIdx.x;
   int a = data[tid + 1];
__syncthreads();    // redundant
int b = data[tid + 1];

}

__global__ void test4(int *data) {
    int tid = threadIdx.x;
   int a = data[tid + 1];
__syncthreads();   // redundant
data[tid + 1] = tid;

}
__global__ void test5(int *data) {
    int tid = threadIdx.x;
    data[tid + 1] = tid;
__syncthreads();    // redundant
int b = data[tid + 1];

}

__global__ void test6(int *data) {
    int tid = threadIdx.x;
   data[tid + 1] = 1;
__syncthreads();    // redundant
data[tid + 1] = 2;
}


__global__ void test7(int *data) {
    int tid = threadIdx.x;
   int a = data[tid +1];
__syncthreads();    // redundant
int b = data[2*tid + 1];

}

__global__ void test8(int *data) {
    int tid = threadIdx.x;
   int a = data[tid + 1];
__syncthreads();    // redundant
data[2*tid + 1] = tid;

}
__global__ void test9(int *data) {
    int tid = threadIdx.x;
    data[tid + 1] = tid;
__syncthreads();    // neccesary
int b = data[2*tid + 1];

}

__global__ void test10(int *data) {
    int tid = threadIdx.x;
   data[tid + 1] = 1;
__syncthreads();    // neccesary
data[2*tid + 1] = 2;
}


__global__ void test11(int *data,int *data2) {
    int tid = threadIdx.x;
    data[tid] = tid;
    data2[tid] = tid + 1;
    __syncthreads();  // Necessary: data2 is written by other threads
    int val = data2[tid];
    int val2 = data[tid+1];
    __syncthreads();  // neccesary: data is written by other threads
    data[tid] = val * 2;
}

int main() {
    int *d_data;
    hipMalloc(&d_data, sizeof(int) * 256);
    test1<<<1, 256>>>(d_data);
    test2<<<1, 256>>>(d_data);
    test3<<<1, 256>>>(d_data);
    test4<<<1, 256>>>(d_data);
    test5<<<1, 256>>>(d_data);  
    test6<<<1, 256>>>(d_data);
    test7<<<1, 256>>>(d_data);
    test8<<<1, 256>>>(d_data);
    test9<<<1, 256>>>(d_data);
    test10<<<1, 256>>>(d_data);
    int *d_data2;
    hipMalloc(&d_data2, sizeof(int) * 256);
    test11<<<1, 256>>>(d_data, d_data2);
    hipDeviceSynchronize();
    hipFree(d_data);
    return 0;
}
