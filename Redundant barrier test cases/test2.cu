
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void test(int *data) {
    int tid = threadIdx.x;
    data[3]= 1;
    __syncthreads();  // Redundant: no access after this depends on data written before
    data[1] = tid;
    
    __syncthreads();  // Redundant: no access after this depends on data written before
    int a= data[2];
    __syncthreads();  // Redundant: no access after this depends on data written before
    int b= data[3];
        
}


int main() {
    int *d_data;
    hipMalloc(&d_data, sizeof(int) * 256);
    test<<<1, 256>>>(d_data);
    hipDeviceSynchronize();
    hipFree(d_data);
    return 0;
}
